
#include <hip/hip_runtime.h>

#ifdef WHYDRO2

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "prototypes.h"
#include <mpi.h>
#include <cudpp.h>

#define NTHREAD 128
#define NITERMAX 10
#define ERRTOL 1e-10


extern "C" struct OCT *gatherstencil(struct OCT *octstart, struct HGRID *stencil, int stride, struct CPUINFO *cpu, int *nread);
extern "C" struct OCT *scatterstencil(struct OCT *octstart, struct HGRID *stencil, int stride, struct CPUINFO *cpu, REAL dxcur, REAL dtnew);
extern "C" void create_hydstencil_GPU(struct CPUINFO *cpu, int stride);
extern "C" int advancehydroGPU(struct OCT **firstoct, int level, struct CPUINFO *cpu, struct HGRID *stencil, int stride, REAL dxcur, REAL dtnew);
extern "C" void create_pinned_stencil(struct HGRID **stencil, int stride);
extern "C" void destroy_hydstencil_GPU(struct CPUINFO *cpu, int stride);
extern "C" void destroy_pinned_stencil(struct HGRID **stencil, int stride);

// ===================================================================
void create_hydstencil_GPU(struct CPUINFO *cpu, int stride){
  cudaMalloc((void **)&(cpu->hyd_stencil),sizeof(struct HGRID)*stride);
}

// ===================================================================
void create_pinned_stencil(struct HGRID **stencil, int stride){
  cudaMallocHost( (void**)stencil, sizeof(struct HGRID)*stride );
}

// ===================================================================
void destroy_hydstencil_GPU(struct CPUINFO *cpu, int stride){
  cudaFree(cpu->hyd_stencil);
}

// ===================================================================
void destroy_pinned_stencil(struct HGRID **stencil, int stride){
  cudaFreeHost(stencil);
}



// ==============================================================================================================

__device__ void dgetE(struct Wtype *W){
  W->E=W->p/(GAMMA-1.)+0.5*W->d*(W->u*W->u+W->v*W->v+W->w*W->w);
}

// =======================================================

__device__ void getcellnei_gpu_hydro(int cindex, int *neip, int *cell)
{
  switch(cindex){
  case 0:
    neip[0]=0;cell[0]=1;
    neip[1]=6;cell[1]=1;
    neip[2]=2;cell[2]=2;
    neip[3]=6;cell[3]=2;
    neip[4]=4;cell[4]=4;
    neip[5]=6;cell[5]=4;
    break;
  case 1:
    neip[0]=6;cell[0]=0;
    neip[1]=1;cell[1]=0;
    neip[2]=2;cell[2]=3;
    neip[3]=6;cell[3]=3;
    neip[4]=4;cell[4]=5;
    neip[5]=6;cell[5]=5;
    break;
  case 2:
    neip[0]=0;cell[0]=3;
    neip[1]=6;cell[1]=3;
    neip[2]=6;cell[2]=0;
    neip[3]=3;cell[3]=0;
    neip[4]=4;cell[4]=6;
    neip[5]=6;cell[5]=6;
    break;
  case 3:
    neip[0]=6;cell[0]=2;
    neip[1]=1;cell[1]=2;
    neip[2]=6;cell[2]=1;
    neip[3]=3;cell[3]=1;
    neip[4]=4;cell[4]=7;
    neip[5]=6;cell[5]=7;
    break;
  case 4:
    neip[0]=0;cell[0]=5;
    neip[1]=6;cell[1]=5;
    neip[2]=2;cell[2]=6;
    neip[3]=6;cell[3]=6;
    neip[4]=6;cell[4]=0;
    neip[5]=5;cell[5]=0;
    break;
  case 5:
    neip[0]=6;cell[0]=4;
    neip[1]=1;cell[1]=4;
    neip[2]=2;cell[2]=7;
    neip[3]=6;cell[3]=7;
    neip[4]=6;cell[4]=1;
    neip[5]=5;cell[5]=1;
    break;
  case 6:
    neip[0]=0;cell[0]=7;
    neip[1]=6;cell[1]=7;
    neip[2]=6;cell[2]=4;
    neip[3]=3;cell[3]=4;
    neip[4]=6;cell[4]=2;
    neip[5]=5;cell[5]=2;
    break;
  case 7:
    neip[0]=6;cell[0]=6;
    neip[1]=1;cell[1]=6;
    neip[2]=6;cell[2]=5;
    neip[3]=3;cell[3]=5;
    neip[4]=6;cell[4]=3;
    neip[5]=5;cell[5]=3;
    break;
  }

}

// ==================== converts U -> W
__device__ void dU2W(struct Utype *U, struct Wtype *W)
{
  W->d=U->d;
  W->u=U->du/U->d;
  W->v=U->dv/U->d;
  W->w=U->dw/U->d;
  
#ifdef DUAL_E
  W->p=U->eint*(GAMMA-1.);
  W->E=U->E;
#ifdef WRADHYD
  W->X=U->dX/U->d;
#endif
#else
  W->p=(GAMMA-1.)*(U->E-((U->du)*(U->du)+(U->dv)*(U->dv)+(U->dw)*(U->dw))/(U->d)*0.5);
#endif
  W->a=sqrt(GAMMA*W->p/W->d);
}

// ==================== converts W -> U
__device__ void dW2U(struct Wtype *W, struct Utype *U)
{
  U->d=W->d;
  U->du=W->d*W->u;
  U->dv=W->d*W->v;
  U->dw=W->d*W->w;

#ifdef DUAL_E
  U->eint=W->p/(GAMMA-1.);
  U->E=W->E;
#ifdef WRADHYD
  U->dX=W->d*W->X;
#endif
#endif

}



// ---------------------------------------------------------------
__device__ void dgetflux_X(struct Utype *U, REAL *f)
{
  f[0]=U->du;
  f[1]=0.5*(3.-GAMMA)*U->du*U->du/U->d+(GAMMA-1.)*U->E-0.5*(GAMMA-1.)*(U->dv*U->dv+U->dw*U->dw)/U->d;
  f[2]=U->du*U->dv/U->d;
  f[3]=U->du*U->dw/U->d;
  f[4]=GAMMA*U->du/U->d*U->E-0.5*(GAMMA-1.)*U->du/(U->d*U->d)*(U->du*U->du+U->dv*U->dv+U->dw*U->dw);
#ifdef WRADHYD
  f[6]=U->du*U->dX/U->d;
#endif
}

// ---------------------------------------------------------------

__device__ void dgetflux_Y(struct Utype *U, REAL *f)
{
  f[0]=U->dv;
  f[1]=U->dv*U->du/U->d;
  f[2]=0.5*(3.-GAMMA)*U->dv*U->dv/U->d+(GAMMA-1.)*U->E-0.5*(GAMMA-1.)*(U->du*U->du+U->dw*U->dw)/U->d;
  f[3]=U->dv*U->dw/U->d;
  f[4]=GAMMA*U->dv/U->d*U->E-0.5*(GAMMA-1.)*U->dv/(U->d*U->d)*(U->du*U->du+U->dv*U->dv+U->dw*U->dw);
#ifdef WRADHYD
  f[6]=U->dv*U->dX/U->d;
#endif
}

// ---------------------------------------------------------------

__device__ void dgetflux_Z(struct Utype *U, REAL *f)
{
  f[0]=U->dw;
  f[1]=U->dw*U->du/U->d;
  f[2]=U->dw*U->dv/U->d;
  f[3]=0.5*(3.-GAMMA)*U->dw*U->dw/U->d+(GAMMA-1.)*U->E-0.5*(GAMMA-1.)*(U->du*U->du+U->dv*U->dv)/U->d;
  f[4]=GAMMA*U->dw/U->d*U->E-0.5*(GAMMA-1.)*U->dw/(U->d*U->d)*(U->du*U->du+U->dv*U->dv+U->dw*U->dw);
#ifdef WRADHYD
  f[6]=U->dw*U->dX/U->d;
#endif
}



// ================== performs the difference between two Us

__device__ void ddiffU(struct Utype *U2, struct Utype *U1, struct Utype *UR){
  
  UR->d =U2->d - U1->d;
  UR->du=U2->du- U1->du;
  UR->dv=U2->dv- U1->dv;
  UR->dw=U2->dw- U1->dw;
  UR->E =U2->E - U1->E;
#ifdef DUAL_E
  UR->eint=U2->eint-U1->eint;
#endif
}

// ================== performs the difference between two Ws

__device__ void ddiffW(struct Wtype *W2, struct Wtype *W1, struct Wtype *WR){

  WR->d=W2->d- W1->d;
  WR->u=W2->u- W1->u;
  WR->v=W2->v- W1->v;
  WR->w=W2->w- W1->w;
  WR->p=W2->p- W1->p;
}




// ================= minmod
__device__ void dminmod(struct Utype *Um, struct Utype *Up, struct Utype *Ur){

  REAL beta=1.; // 1. for MINBEE 2. for SUPERBEE
  // FLUX LIMITER

  if(Up->d>0){
    Ur->d=fmax(fmax(0.,fmin(beta*Um->d,Up->d)),fmin(Um->d,beta*Up->d));
  }
  else{
    Ur->d=fmin(fmin(0.,fmax(beta*Um->d,Up->d)),fmax(Um->d,beta*Up->d));
  }


  if(Up->du>0){
    Ur->du=fmax(fmax(0.,fmin(beta*Um->du,Up->du)),fmin(Um->du,beta*Up->du));
  }
  else{
    Ur->du=fmin(fmin(0.,fmax(beta*Um->du,Up->du)),fmax(Um->du,beta*Up->du));
  }


  if(Up->dv>0){
    Ur->dv=fmax(fmax(0.,fmin(beta*Um->dv,Up->dv)),fmin(Um->dv,beta*Up->dv));
  }
  else{
    Ur->dv=fmin(fmin(0.,fmax(beta*Um->dv,Up->dv)),fmax(Um->dv,beta*Up->dv));
  }


  if(Up->dw>0){
    Ur->dw=fmax(fmax(0.,fmin(beta*Um->dw,Up->dw)),fmin(Um->dw,beta*Up->dw));
  }
  else{
    Ur->dw=fmin(fmin(0.,fmax(beta*Um->dw,Up->dw)),fmax(Um->dw,beta*Up->dw));
  }


  if(Up->E>0){
    Ur->E=fmax(fmax(0.,fmin(beta*Um->E,Up->E)),fmin(Um->E,beta*Up->E));
  }
  else{
    Ur->E=fmin(fmin(0.,fmax(beta*Um->E,Up->E)),fmax(Um->E,beta*Up->E));
  }


}

//===============================================
//===============================================

__device__ void dminmod_W(struct Wtype *Wm, struct Wtype *Wp, struct Wtype *Wr){

  REAL beta=1.; // 1. for MINBEE 2. for SUPERBEE
  // FLUX LIMITER

  if(Wp->d>0){
    Wr->d=fmax(fmax(0.,fmin(beta*Wm->d,Wp->d)),fmin(Wm->d,beta*Wp->d));
  }
  else{
    Wr->d=fmin(fmin(0.,fmax(beta*Wm->d,Wp->d)),fmax(Wm->d,beta*Wp->d));
  }


  if(Wp->u>0){
    Wr->u=fmax(fmax(0.,fmin(beta*Wm->u,Wp->u)),fmin(Wm->u,beta*Wp->u));
  }
  else{
    Wr->u=fmin(fmin(0.,fmax(beta*Wm->u,Wp->u)),fmax(Wm->u,beta*Wp->u));
  }


  if(Wp->v>0){
    Wr->v=fmax(fmax(0.,fmin(beta*Wm->v,Wp->v)),fmin(Wm->v,beta*Wp->v));
  }
  else{
    Wr->v=fmin(fmin(0.,fmax(beta*Wm->v,Wp->v)),fmax(Wm->v,beta*Wp->v));
  }


  if(Wp->w>0){
    Wr->w=fmax(fmax(0.,fmin(beta*Wm->w,Wp->w)),fmin(Wm->w,beta*Wp->w));
  }
  else{
    Wr->w=fmin(fmin(0.,fmax(beta*Wm->w,Wp->w)),fmax(Wm->w,beta*Wp->w));
  }


  if(Wp->p>0){
    Wr->p=fmax(fmax(0.,fmin(beta*Wm->p,Wp->p)),fmin(Wm->p,beta*Wp->p));
  }
  else{
    Wr->p=fmin(fmin(0.,fmax(beta*Wm->p,Wp->p)),fmax(Wm->p,beta*Wp->p));
  }


}


// ============= interp minmod ====================================================

__device__ void dinterpminmod(struct Utype *U0, struct Utype *Up, struct Utype *Dx, struct Utype *Dy, struct Utype *Dz,REAL dx,REAL dy,REAL dz){
  
  Up->d =U0->d  + dx*Dx->d  +dy*Dy->d  +dz*Dz->d;
  Up->du=U0->du + dx*Dx->du +dy*Dy->du +dz*Dz->du;
  Up->dv=U0->dv + dx*Dx->dv +dy*Dy->dv +dz*Dz->dv;
  Up->dw=U0->dw + dx*Dx->dw +dy*Dy->dw +dz*Dz->dw;
  Up->E =U0->E  + dx*Dx->E  +dy*Dy->E  +dz*Dz->E;
#ifdef DUAL_E
  Up->eint =U0->eint  + dx*Dx->eint  +dy*Dy->eint  +dz*Dz->eint;
#endif

}

// ============= interp minmod ====================================================

__device__ void dinterpminmod_W(struct Wtype *W0, struct Wtype *Wp, struct Wtype *Dx, struct Wtype *Dy, struct Wtype *Dz,REAL dx,REAL dy,REAL dz){
  
  Wp->d =W0->d +dx*Dx->d +dy*Dy->d +dz*Dz->d;
  Wp->u =W0->u +dx*Dx->u +dy*Dy->u +dz*Dz->u;
  Wp->v =W0->v +dx*Dx->v +dy*Dy->v +dz*Dz->v;
  Wp->w =W0->w +dx*Dx->w +dy*Dy->w +dz*Dz->w;
  Wp->p =W0->p +dx*Dx->p +dy*Dy->p +dz*Dz->p;

}

//========================================================================================================================================

__device__ void  dmatrix_jacobian(struct Wtype *W0, REAL dt,REAL dx,struct Wtype *Dx,struct Wtype *Dy,struct Wtype *Dz, struct Wtype *Wt){


  REAL M[25];
  REAL W[5]={0.,0.,0.,0.,0.};
  REAL d[5];
  int i,j;


  // =====  building the A matrix

  memset(M,0,25*sizeof(REAL));
  
  // diagonal elements
  for(i=0;i<5;i++) M[i+i*5]=W0->u;
  
  // off_diagonal elements
  M[0+1*5]=W0->d;

  M[4+1*5]=W0->d*W0->a*W0->a;

  M[1+4*5]=1./W0->d;


  // ===== First Product

  d[0]=Dx->d;
  d[1]=Dx->u;
  d[2]=Dx->v;
  d[3]=Dx->w;
  d[4]=Dx->p;

  for(j=0;j<5;j++){
    for(i=0;i<5;i++){
      W[i]+=M[i+j*5]*d[j];
      }
  }

  // =====  building the B matrix

  memset(M,0,25*sizeof(REAL));
  
  // diagonal elements
  for(i=0;i<5;i++) M[i+i*5]=W0->v;
  
  // off_diagonal elements
  M[0+2*5]=W0->d;

  M[4+2*5]=W0->d*W0->a*W0->a;

  M[2+4*5]=1./W0->d;


  // ===== Second Product

  d[0]=Dy->d;
  d[1]=Dy->u;
  d[2]=Dy->v;
  d[3]=Dy->w;
  d[4]=Dy->p;

  for(j=0;j<5;j++){
    for(i=0;i<5;i++){
      W[i]+=M[i+j*5]*d[j];
      }
  }

  // =====  building the C matrix

  memset(M,0,25*sizeof(REAL));
  
  // diagonal elements
  for(i=0;i<5;i++) M[i+i*5]=W0->w;
  
  // off_diagonal elements
  M[0+3*5]=W0->d;

  M[4+3*5]=W0->d*W0->a*W0->a;

  M[3+4*5]=1./W0->d;

  d[0]=Dz->d;
  d[1]=Dz->u;
  d[2]=Dz->v;
  d[3]=Dz->w;
  d[4]=Dz->p;

 
  for(j=0;j<5;j++){
    for(i=0;i<5;i++){
      W[i]+=M[i+j*5]*d[j];
      }
  }
  
  // ==== Final correction
  for(i=0;i<5;i++){
    W[i]*=(-dt/dx*0.5);
  }
  
  Wt->d=W[0];
  Wt->u=W[1];
  Wt->v=W[2];
  Wt->w=W[3];
  Wt->p=W[4];
  
}

// ==============================================
__device__ void dMUSCL_BOUND2(struct HGRID *stencil, int ioct, int icell, struct Wtype *Wi,REAL dt,REAL dx){ 
  
  struct Wtype *W0;
  struct Wtype *Wp;
  struct Wtype *Wm;
  struct Wtype Dp,Dm;
  struct Wtype D[3];
  struct Wtype Wt;
  int inei2;
  int vcell[6],vnei[6];
  int dir;
  int idir;
  int shift;

#ifdef WGRAV
	  REAL f[3];
#ifdef CONSERVATIVE
	  struct Utype S;
	  struct Utype U;
#endif
#endif

	  getcellnei_gpu_hydro(icell, vnei, vcell); // we get the neighbors
	  
	  W0=&(stencil->oct[ioct].cell[icell].field);
	
	  // Limited Slopes
	  shift=1;
	  for(dir=0;dir<3;dir++){
	    
	    inei2=2*dir;
	    if(vnei[inei2]==6){
	      Wm=&(stencil->oct[ioct].cell[vcell[inei2]].field);
	    }
	    else{
	      Wm=&(stencil->oct[ioct-shift].cell[vcell[inei2]].field);
	    }

	    inei2=2*dir+1;
	    if(vnei[inei2]==6){
	      Wp=&(stencil->oct[ioct].cell[vcell[inei2]].field);
	    }
	    else{
	      Wp=&(stencil->oct[ioct+shift].cell[vcell[inei2]].field);
	    }

	    ddiffW(Wp,W0,&Dp); 
	    ddiffW(W0,Wm,&Dm); 
	    
	    dminmod_W(&Dm,&Dp,D+dir);
	    shift*=3;
	  }


	  // build jacobian matrix product
	  
	  dmatrix_jacobian(W0,dt,dx,&D[0],&D[1],&D[2],&Wt); // Here Wt contains the evolution of the state
	  
	  // READY TO EVOLVE EXTRAPOLATED VALUE

	  REAL ix[]={-0.5,0.5,0.0,0.0,0.0,0.0};
	  REAL iy[]={0.0,0.0,-0.5,0.5,0.0,0.0};
	  REAL iz[]={0.0,0.0,0.0,0.0,-0.5,0.5};

#ifdef WGRAV
#ifndef NOCOUPLE
	  f[0]=stencil->oct[ioct].cell[icell].f[0];
	  f[1]=stencil->oct[ioct].cell[icell].f[1];
	  f[2]=stencil->oct[ioct].cell[icell].f[2];
#ifdef CONSERVATIVE
	  S.d =0.;
	  S.du=-W0->d*f[0]*0.5*dt;
	  S.dv=-W0->d*f[1]*0.5*dt;
	  S.dw=-W0->d*f[2]*0.5*dt;
	  S.E =-(W0->d*W0->u*f[0]+W0->d*W0->v*f[1]+W0->d*W0->w*f[2])*dt*0.5;
#endif

#endif
#endif
	  for(idir=0;idir<6;idir++){
	    Wi[idir].d = W0->d+ix[idir]*D[0].d+iy[idir]*D[1].d+iz[idir]*D[2].d+Wt.d;
	    Wi[idir].u = W0->u+ix[idir]*D[0].u+iy[idir]*D[1].u+iz[idir]*D[2].u+Wt.u;
	    Wi[idir].v = W0->v+ix[idir]*D[0].v+iy[idir]*D[1].v+iz[idir]*D[2].v+Wt.v;
	    Wi[idir].w = W0->w+ix[idir]*D[0].w+iy[idir]*D[1].w+iz[idir]*D[2].w+Wt.w;
	    Wi[idir].p = fmax(W0->p+ix[idir]*D[0].p+iy[idir]*D[1].p+iz[idir]*D[2].p+Wt.p,PMIN);


	    /* if(Wi[idir].p<0) abort(); */
	    /* if(Wi[idir].d<0) abort(); */


#ifdef WGRAV
#ifndef NOCOUPLE

#ifdef PRIMITIVE
	    Wi[idir].u+=-f[0]*0.5*dt;
	    Wi[idir].v+=-f[1]*0.5*dt;
	    Wi[idir].w+=-f[2]*0.5*dt;
#endif

#ifdef CONSERVATIVE
 	    dW2U(&Wi[idir],&U);
	    U.d  +=S.d;
	    U.du +=S.du;
	    U.dv +=S.dv;
	    U.dw +=S.dw;
	    U.E  +=S.E;
	    dU2W(&U,&Wi[idir]);
#endif

#endif
#endif
	    
	    //if(Wi[idir].p<0) abort();
	    //Wi[idir].E=Wi[idir].p/(GAMMA-1.)+0.5*Wi[idir].d*(Wi[idir].u*Wi[idir].u+Wi[idir].v*Wi[idir].v+Wi[idir].w*Wi[idir].w);
	    dgetE(Wi+idir);
	    Wi[idir].a=sqrt(GAMMA*Wi[idir].p/Wi[idir].d);

#ifdef WRADHYD
	    Wi[idir].X=W0->X;
#endif
	  }



	  
}

//========================================================================================
__device__ REAL dfrootprime(REAL p, struct Wtype1D *WL, struct Wtype1D *WR)
{
  
  REAL fL,fR;
  REAL AL,AR,BL,BR;

  AL=2./((GAMMA+1.)*WL->d);
  AR=2./((GAMMA+1.)*WR->d);
  
  BL=(GAMMA-1.)/(GAMMA+1.)*WL->p;
  BR=(GAMMA-1.)/(GAMMA+1.)*WR->p;

  fL=(p>WL->p?sqrt(AL/(BL+p))*(1.-(p-WL->p)/(2.*(BL+p))):pow(p/WL->p,-(GAMMA+1)/(2.*GAMMA))/(WL->d*WL->a));
  fR=(p>WR->p?sqrt(AR/(BR+p))*(1.-(p-WR->p)/(2.*(BR+p))):pow(p/WR->p,-(GAMMA+1)/(2.*GAMMA))/(WR->d*WR->a));

  return fL+fR;
}


// ------------------------------------

__device__ REAL dfroot(REAL p, struct Wtype1D *WL, struct Wtype1D *WR, REAL *u)
{
  
  REAL fL,fR;
  REAL AL,AR,BL,BR;
  REAL Deltau;

  AL=2./((GAMMA+1.)*WL->d);
  AR=2./((GAMMA+1.)*WR->d);
  
  BL=(GAMMA-1.)/(GAMMA+1.)*WL->p;
  BR=(GAMMA-1.)/(GAMMA+1.)*WR->p;

  fL=(p>WL->p?(p-WL->p)*sqrt(AL/(BL+p)):2.*WL->a/(GAMMA-1.)*(pow(p/WL->p,(GAMMA-1)/(2.*GAMMA))-1.));
  fR=(p>WR->p?(p-WR->p)*sqrt(AR/(BR+p)):2.*WR->a/(GAMMA-1.)*(pow(p/WR->p,(GAMMA-1)/(2.*GAMMA))-1.));
  
  Deltau=WR->u-WL->u;
  *u=0.5*(WL->u+WR->u)+0.5*(fR-fL);

  return fL+fR+Deltau;
}


//========================================================================================
//========================================================================================
__device__ REAL dfindPressure(struct Wtype1D *WL, struct Wtype1D *WR, int *niter, REAL *u)
{

  double ptr,pts,ppv;
  double ptr0,pts0,ppv0;
  double p,porg,dp;
  int i;
  double err;
  double unsurz=(2.0*GAMMA)/(GAMMA-1.0);
  double AL,AR,BL,BR,GL,GR;
  double pmin,pmax;
  double u2;

  pmin=fmin(WL->p,WR->p);
  pmax=fmax(WL->p,WR->p);
  
  // EXACT SOLVER

  // hybrid guess for pressure

  AL=2./((GAMMA+1.)*WL->d);
  AR=2./((GAMMA+1.)*WR->d);
  
  BL=(GAMMA-1.)/(GAMMA+1.)*WL->p;
  BR=(GAMMA-1.)/(GAMMA+1.)*WR->p;

  ppv0=0.5*(WL->p+WR->p)-0.125*(WR->u-WL->u)*(WR->d+WL->d)*(WR->a+WL->a);
  ptr0=pow((WL->a+WR->a-0.5*(GAMMA-1)*(WR->u-WL->u))/(WL->a/pow(WL->p,1./unsurz)+WR->a/pow(WR->p,1./unsurz)),unsurz);

  ppv=fmax(ERRTOL,ppv0);
  ptr=fmax(ERRTOL,ptr0);
  
  GL=sqrt(AL/(ppv+BL));
  GR=sqrt(AR/(ppv+BR));

  pts0=(GL*WL->p+GR*WR->p-(WR->u-WL->u))/(GL+GR);
  pts=fmax(ERRTOL,pts0);


  if(((pmax/pmin)<2.0)&&((pmin<=ppv)&&(ppv<=pmax))){
      p=ppv;
    }
  else{
    if(ppv<pmin){
      p=ptr;
    }
    else{
      p=pts;
    }
  }


  //p=0.5*(WL->p+WR->p);
  //p=fmax(p,ERRTOL);

  *niter=0;
  for(i=0;i<NITERMAX;i++)
    {
      dp=dfroot(p,WL,WR,&u2)/dfrootprime(p,WL,WR);
      if((isnan(dp))){
      	/* printf("froot=%e frootprime=%e\n",froot(p,WL,WR,&u2),frootprime(p,WL,WR)); */
      	/* abort(); */
      }
      
      if(fabs(dp)<ERRTOL) break;
      while((p-dp)<0){ 
       	dp=dp*0.5; 
      } 

      porg=p;
      p=p-dp;
      //if(frootprime(p,WL,WR)==0) abort();//printf("p0=%e dp=%e p=%e fprime=%e\n",porg,dp,p,frootprime(p,WL,WR));
      err=2.*fabs(p-porg)/(fabs(p+porg));
      *niter=*niter+1;
      //if(p<=0) p=ERRTOL;
      if(err<ERRTOL) break;
      if(dfroot(p,WL,WR,&u2)<ERRTOL) break;
    }

  if(i==NITERMAX){
    //printf("DIVERGENCE p0=%e dp=%e p=%e fprime=%e err=%e\n",porg,dp,p,frootprime(p,WL,WR),err);
    //abort();
  }

  /* if(p>6.4e-7){ */
  /*   printf("MAX p0=%e dp=%e p=%e fprime=%e err=%e\n",porg,dp,p,frootprime(p,WL,WR),err); */
  /*   abort(); */
  /* } */
  dfroot(p,WL,WR,&u2); // last calculation to get u;

  *u=(REAL)u2;
  return p;
}


//========================================================================================
//========================================================================================
__device__ REAL dfindPressure_Hybrid(struct Wtype1D *WL, struct Wtype1D *WR, int *niter, REAL *ustar){
  double ppvrs;
  double dbar,abar;
  double pmax,pmin,pstar;
  double AL,AR,BL,BR,GL,GR;
  dbar=0.5*(WL->d+WR->d);
  abar=0.5*(WL->a+WR->a);
  ppvrs=0.5*((WL->p+WR->p)+(WL->u-WR->u)*dbar*abar);
  pmax=fmax(WL->p,WR->p);
  pmin=fmin(WL->p,WR->p);
  pstar=ppvrs;
  
  if(((pmax/pmin)<2.)&&((pmin<pstar)&&(pstar<pmax))){
    // PVRS CASE
    pstar=ppvrs;
    *ustar=0.5*((WL->u+WR->u)+(WL->p-WR->p)/(dbar*abar));
  }
  else{
    if(pstar<pmin){
      //TRRS CASE
      double z=(GAMMA-1.)/(2.*GAMMA);
      double iz=(2.*GAMMA)/(GAMMA-1.);
      pstar=pow((WL->a+WR->a-(GAMMA-1.)/2.*(WR->u-WL->u))/(WL->a/pow(WL->p,z)+WR->a/pow(WR->p,z)),iz);
      *ustar=WL->u-2.*WL->a/(GAMMA-1.)*(pow(pstar/WL->p,z)-1.);
    }
    else{
      //TSRS CASE
      double p0;
      p0=fmax(0.,ppvrs);
      
      AL=2./((GAMMA+1.)*WL->d);
      AR=2./((GAMMA+1.)*WR->d);
      
      BL=(GAMMA-1.)/(GAMMA+1.)*WL->p;
      BR=(GAMMA-1.)/(GAMMA+1.)*WR->p;

      GL=sqrt(AL/(p0+BL));
      GR=sqrt(AR/(p0+BR));

      pstar=(GL*WL->p+GR*WR->p-(WR->u-WL->u))/(GL+GR);
      *ustar=0.5*((WL->u+WR->u)+(pstar-WR->p)*GR-(pstar-WL->p)*GL);
    }
  }

  return pstar;

}




//====================================================================
__device__ void dspeedestimateX_HLLC(struct Wtype *WL,struct Wtype *WR, REAL *SL, REAL *SR, REAL *pstar, REAL *ustar){

  REAL qL,qR;
  struct Wtype1D WLloc;
  struct Wtype1D WRloc;
  int n;

  WLloc.d=WL->d;
  WLloc.u=WL->u;
  WLloc.p=WL->p;
  WLloc.a=sqrt(GAMMA*WLloc.p/WLloc.d);
  
  WRloc.d=WR->d;
  WRloc.u=WR->u;
  WRloc.p=WR->p;
  WRloc.a=sqrt(GAMMA*WRloc.p/WRloc.d);

  (*pstar)= dfindPressure_Hybrid(&WLloc,&WRloc,&n,ustar);
  if((*pstar)<0) (*pstar)=dfindPressure(&WLloc,&WRloc,&n,ustar);
  //if((*pstar)<0) abort();

  qL=(*pstar<=WL->p?1.:sqrt(1.+(GAMMA+1.)/(2.*GAMMA)*((*pstar)/WL->p-1.)));
  qR=(*pstar<=WR->p?1.:sqrt(1.+(GAMMA+1.)/(2.*GAMMA)*((*pstar)/WR->p-1.)));
  
  *SL=WLloc.u-WLloc.a*qL;
  *SR=WRloc.u+WRloc.a*qR;
  if((*SL)>(*SR)){
    (*SL)=fminf(WLloc.u-WLloc.a,WRloc.u-WRloc.a);
    (*SR)=fmaxf(WLloc.u+WLloc.a,WRloc.u+WRloc.a);
  }
  /* if((*SL)>(*SR)) abort(); */
  /* if(isnan(*ustar)) abort(); */
}

//====================================================================

void __device__ dspeedestimateY_HLLC(struct Wtype *WL,struct Wtype *WR, REAL *SL, REAL *SR, REAL *pstar, REAL *ustar){

  REAL qL,qR;
  struct Wtype1D WLloc;
  struct Wtype1D WRloc;
  int n;

  WLloc.d=WL->d;
  WLloc.u=WL->v;
  WLloc.p=WL->p;
  WLloc.a=sqrt(GAMMA*WLloc.p/WLloc.d);
  
  WRloc.d=WR->d;
  WRloc.u=WR->v;
  WRloc.p=WR->p;
  WRloc.a=sqrt(GAMMA*WRloc.p/WRloc.d);

  (*pstar)=dfindPressure_Hybrid(&WLloc,&WRloc,&n,ustar);
  if((*pstar)<0) (*pstar)=dfindPressure(&WLloc,&WRloc,&n,ustar);
  //  if((*pstar)<0) abort();

  qL=(*pstar<=WL->p?1.:sqrt(1.+(GAMMA+1.)/(2.*GAMMA)*((*pstar)/WL->p-1.)));
  qR=(*pstar<=WR->p?1.:sqrt(1.+(GAMMA+1.)/(2.*GAMMA)*((*pstar)/WR->p-1.)));
  
  *SL=WLloc.u-WLloc.a*qL;
  *SR=WRloc.u+WRloc.a*qR;

  if((*SL)>(*SR)){
    (*SL)=fminf(WLloc.u-WLloc.a,WRloc.u-WRloc.a);
    (*SR)=fmaxf(WLloc.u+WLloc.a,WRloc.u+WRloc.a);
    //abort();
  }
  //  if((*SL)>(*SR)) abort();

}



//====================================================================

void __device__ dspeedestimateZ_HLLC(struct Wtype *WL,struct Wtype *WR, REAL *SL, REAL *SR, REAL *pstar, REAL *ustar){

  REAL qL,qR;
  struct Wtype1D WLloc;
  struct Wtype1D WRloc;
  int n;

  WLloc.d=WL->d;
  WLloc.u=WL->w;
  WLloc.p=WL->p;
  WLloc.a=sqrt(GAMMA*WLloc.p/WLloc.d);
  
  WRloc.d=WR->d;
  WRloc.u=WR->w;
  WRloc.p=WR->p;
  WRloc.a=sqrt(GAMMA*WRloc.p/WRloc.d);

  (*pstar)=dfindPressure_Hybrid(&WLloc,&WRloc,&n,ustar);
  if((*pstar)<0) (*pstar)=dfindPressure(&WLloc,&WRloc,&n,ustar);
  //if((*pstar)<0) abort();

  qL=(*pstar<=WL->p?1.:sqrt(1.+(GAMMA+1.)/(2.*GAMMA)*((*pstar)/WL->p-1.)));
  qR=(*pstar<=WR->p?1.:sqrt(1.+(GAMMA+1.)/(2.*GAMMA)*((*pstar)/WR->p-1.)));
  
  *SL=WLloc.u-WLloc.a*qL;
  *SR=WRloc.u+WRloc.a*qR;
  if((*SL)>(*SR)){
    (*SL)=fminf(WLloc.u-WLloc.a,WRloc.u-WRloc.a);
    (*SR)=fmaxf(WLloc.u+WLloc.a,WRloc.u+WRloc.a);
    //abort();
  }
  //if((*SL)>(*SR)) abort();

}


// =============================================================================================

__global__ void dhydroM_sweepZ(struct HGRID *stencil, int nread,REAL dx, REAL dt){

  int inei,icell,iface;
  int i;
  int vnei[6],vcell[6];

  REAL FL[NVAR],FR[NVAR];
  struct Utype Uold;
  struct Wtype Wold;
  REAL pstar,ustar;

  struct Wtype WT[6]; // FOR MUSCL RECONSTRUCTION
  struct Wtype WC[6]; // FOR MUSCL RECONSTRUCTION

  struct Utype UC[2];
  struct Utype UN[2];
  struct Wtype WN[2];

  int ioct[7]={12,14,10,16,4,22,13};
  int idxnei[6]={1,0,3,2,5,4};

  struct Wtype *curcell;

  REAL SL,SR;
  
  int ffact[2]={0,0};
  REAL fact;

#ifdef DUAL_E
  struct Utype Us;
  REAL ebar;
  REAL divu,divuloc;
#endif
  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;

  i=bx*blockDim.x+tx;
  if(i<nread){
  for(icell=0;icell<8;icell++){ // we scan the cells
    getcellnei_gpu_hydro(icell, vnei, vcell); // we get the neighbors
      
      
      memset(FL,0,sizeof(REAL)*NVAR);
      memset(FR,0,sizeof(REAL)*NVAR);

      // Getting the original state ===========================
      
      curcell=&(stencil[i].oct[ioct[6]].cell[icell].field);

#ifdef DUAL_E
      divu=stencil[i].New.cell[icell].divu;
#endif      

      Wold.d=curcell->d;
      Wold.u=curcell->u;
      Wold.v=curcell->v;
      Wold.w=curcell->w;
      Wold.p=curcell->p;
      Wold.a=sqrt(GAMMA*Wold.p/Wold.d);

#ifdef WRADHYD
      Wold.X=curcell->X;
#endif

      dW2U(&Wold,&Uold); // primitive -> conservative

      REAL eold=Uold.eint;

      /* // MUSCL STATE RECONSTRUCTION */
      memset(ffact,0,sizeof(int)*2);

      dMUSCL_BOUND2(stencil+i, 13, icell, WC,dt,dx);// central

      for(iface=0;iface<2;iface++){
	inei=iface+4;
	memcpy(WC+iface,WC+inei,sizeof(struct Wtype)); // moving the data towards idx=0,1
	dW2U(WC+iface,UC+iface);
      }

      // Neighbor MUSCL reconstruction
      for(iface=0;iface<2;iface++){
	inei=iface+4;
	dMUSCL_BOUND2(stencil+i, ioct[vnei[inei]], vcell[inei], WT,dt,dx);// 
	memcpy(WN+iface,WT+idxnei[inei],sizeof(struct Wtype)); 
	//memcpy(WN+iface,&(stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].field),sizeof(struct Wtype)); 
	dW2U(WN+iface,UN+iface);
	
	if(!stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].split){
	  ffact[iface]=1; // we cancel the contriubtion of split neighbors
	}

      }


      // Z DIRECTION =========================================================================
      
      // --------- solving the Riemann Problems BOTTOM

      // Switching to Split description

      /* 	// =========================================== */

#ifdef RIEMANN_HLLC
      dspeedestimateZ_HLLC(&WN[0],&WC[0],&SL,&SR,&pstar,&ustar);

      if(SL>=0.){
	dgetflux_Z(&UN[0],FL);
#ifdef DUAL_E
	memcpy(&Us,&UN[0],sizeof(struct Utype));
#endif

      }
      else if(SR<=0.){
	dgetflux_Z(&UC[0],FL);
#ifdef DUAL_E
	memcpy(&Us,&UC[0],sizeof(struct Utype));
#endif
      }
      else if((SL<0.)&&(ustar>=0.)){
	dgetflux_Z(&UN[0],FL);
	fact=WN[0].d*(SL-WN[0].w)/(SL-ustar);
	FL[0]+=(fact*1.                                                                      -UN[0].d )*SL;
	FL[1]+=(fact*WN[0].u                                                                 -UN[0].du)*SL;
	FL[2]+=(fact*WN[0].v                                                                 -UN[0].dv)*SL;
	FL[3]+=(fact*ustar                                                                   -UN[0].dw)*SL;
	FL[4]+=(fact*(UN[0].E/UN[0].d+(ustar-WN[0].w)*(ustar+WN[0].p/(WN[0].d*(SL-WN[0].w))))-UN[0].E )*SL;

#ifdef DUAL_E
	Us.d =(fact*1.);
	Us.du=(fact*WN[0].u);
	Us.dv=(fact*WN[0].v);
	Us.dw=(fact*ustar);
	Us.E =(fact*(UN[0].E/UN[0].d+(ustar-WN[0].w)*(ustar+WN[0].p/(WN[0].d*(SL-WN[0].w)))));
#endif
	
#ifdef WRADHYD
	FL[6]+=(fact*WN[0].X                                                                 -UN[0].dX)*SL;
#endif

      }
      else if((ustar<=0.)&&(SR>0.)){
	dgetflux_Z(&UC[0],FL);
	fact=WC[0].d*(SR-WC[0].w)/(SR-ustar);
	FL[0]+=(fact*1.                                                                      -UC[0].d )*SR;
	FL[1]+=(fact*WC[0].u                                                                 -UC[0].du)*SR;
	FL[2]+=(fact*WC[0].v                                                                 -UC[0].dv)*SR;
	FL[3]+=(fact*ustar                                                                   -UC[0].dw)*SR;
	FL[4]+=(fact*(UC[0].E/UC[0].d+(ustar-WC[0].w)*(ustar+WC[0].p/(WC[0].d*(SR-WC[0].w))))-UC[0].E )*SR;

#ifdef DUAL_E
	Us.d =(fact*1.);
	Us.du=(fact*WC[0].u);
	Us.dv=(fact*WC[0].v);
	Us.dw=(fact*ustar);
	Us.E =(fact*(UC[0].E/UC[0].d+(ustar-WC[0].w)*(ustar+WC[0].p/(WC[0].d*(SR-WC[0].w)))));
#endif

#ifdef WRADHYD
	FL[6]+=(fact*WC[0].X                                                                 -UC[0].dX)*SR;
#endif
      }

#ifdef DUAL_E
      ebar=(Us.E-0.5*(Us.du*Us.du+Us.dv*Us.dv+Us.dw*Us.dw)/Us.d); 
      divuloc=(GAMMA-1.)*(Us.dw/Us.d)*eold;
      FL[5]=(Us.dw/Us.d*ebar);
      divu+=-divuloc;
#endif

#endif
      // ===========================================



      // --------- solving the Riemann Problems TOP


      // Switching to Split description

      //=====================================================

#ifdef RIEMANN_HLLC
      dspeedestimateZ_HLLC(&WC[1],&WN[1],&SL,&SR,&pstar,&ustar);

      if(SL>=0.){
	dgetflux_Z(&UC[1],FR);
#ifdef DUAL_E
	memcpy(&Us,&UC[1],sizeof(struct Utype));
#endif

      }
      else if(SR<=0.){
	dgetflux_Z(&UN[1],FR);
#ifdef DUAL_E
	memcpy(&Us,&UN[1],sizeof(struct Utype));
#endif

      }
      else if((SL<0.)&&(ustar>=0.)){
	dgetflux_Z(&UC[1],FR);
	fact=WC[1].d*(SL-WC[1].w)/(SL-ustar);
	FR[0]+=(fact*1.                                                                      -UC[1].d )*SL;
	FR[1]+=(fact*WC[1].u                                                                 -UC[1].du)*SL;
	FR[2]+=(fact*WC[1].v                                                                 -UC[1].dv)*SL;
	FR[3]+=(fact*ustar                                                                   -UC[1].dw)*SL;
	FR[4]+=(fact*(UC[1].E/UC[1].d+(ustar-WC[1].w)*(ustar+WC[1].p/(WC[1].d*(SL-WC[1].w))))-UC[1].E )*SL;

#ifdef DUAL_E
	Us.d =(fact*1.);
	Us.du=(fact*WC[1].u);
	Us.dv=(fact*WC[1].v);
	Us.dw=(fact*ustar);
	Us.E =(fact*(UC[1].E/UC[1].d+(ustar-WC[1].w)*(ustar+WC[1].p/(WC[1].d*(SL-WC[1].w)))));
#endif
#ifdef WRADHYD
	FR[6]+=(fact*WC[1].X                                                                 -UC[1].dX)*SL;
#endif
      }
      else if((ustar<=0.)&&(SR>0.)){
	dgetflux_Z(&UN[1],FR);
	fact=WN[1].d*(SR-WN[1].w)/(SR-ustar);
	FR[0]+=(fact*1.                                                                      -UN[1].d )*SR;
	FR[1]+=(fact*WN[1].u                                                                 -UN[1].du)*SR;
	FR[2]+=(fact*WN[1].v                                                                 -UN[1].dv)*SR;
	FR[3]+=(fact*ustar                                                                   -UN[1].dw)*SR;
	FR[4]+=(fact*(UN[1].E/UN[1].d+(ustar-WN[1].w)*(ustar+WN[1].p/(WN[1].d*(SR-WN[1].w))))-UN[1].E )*SR;

#ifdef DUAL_E
	Us.d =(fact*1.);
	Us.du=(fact*WN[1].u);
	Us.dv=(fact*WN[1].v);
	Us.dw=(fact*ustar);
	Us.E =(fact*(UN[1].E/UN[1].d+(ustar-WN[1].w)*(ustar+WN[1].p/(WN[1].d*(SR-WN[1].w)))));
#endif
	
#ifdef WRADHYD
	FR[6]+=(fact*WN[1].X                                                                 -UN[1].dX)*SR;
#endif
      }

#ifdef DUAL_E
      ebar=(Us.E-0.5*(Us.du*Us.du+Us.dv*Us.dv+Us.dw*Us.dw)/Us.d); 
      divuloc=(GAMMA-1.)*(Us.dw/Us.d)*eold;
      FR[5]=(Us.dw/Us.d*ebar);
      divu+= divuloc;
#endif


#endif


      //========================= copy the fluxes

      // Cancelling the fluxes from splitted neighbours

      for(iface=0;iface<NVAR;iface++) FL[iface]*=ffact[0]; 
      for(iface=0;iface<NVAR;iface++) FR[iface]*=ffact[1]; 


      memcpy(stencil[i].New.cell[icell].flux+4*NVAR,FL,sizeof(REAL)*NVAR);
      memcpy(stencil[i].New.cell[icell].flux+5*NVAR,FR,sizeof(REAL)*NVAR);

      stencil[i].New.cell[icell].divu=divu;

      //ready for the next oct
  }
  }
}




//============================================================================
// =============================================================================================

__global__ void dhydroM_sweepY(struct HGRID *stencil,int nread,REAL dx, REAL dt){

  int inei,icell,iface;
  int i;
  int vnei[6],vcell[6];
  
  REAL FL[NVAR],FR[NVAR];
  struct Utype Uold;
  struct Wtype Wold;
  REAL pstar,ustar;

  struct Wtype WT[6]; // FOR MUSCL RECONSTRUCTION
  struct Wtype WC[6]; // FOR MUSCL RECONSTRUCTION

  struct Utype UC[2];
  struct Utype UN[2];
  struct Wtype WN[2];

  int ioct[7]={12,14,10,16,4,22,13};
  int idxnei[6]={1,0,3,2,5,4};

  struct Wtype *curcell;

  REAL SL,SR;
  
  int ffact[2]={0,0};
  REAL fact;

#ifdef DUAL_E
  struct Utype Us;
  REAL ebar;
  REAL divu,divuloc;
#endif
  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;
	
  i=bx*blockDim.x+tx;
  if(i<nread){
  for(icell=0;icell<8;icell++){ // we scan the cells
    getcellnei_gpu_hydro(icell, vnei, vcell); // we get the neighbors
      
  
      
      memset(FL,0,sizeof(REAL)*NVAR);
      memset(FR,0,sizeof(REAL)*NVAR);

      // Getting the original state ===========================
      
      curcell=&(stencil[i].oct[ioct[6]].cell[icell].field);

#ifdef DUAL_E
      divu=stencil[i].New.cell[icell].divu;
#endif      

      Wold.d=curcell->d;
      Wold.u=curcell->u;
      Wold.v=curcell->v;
      Wold.w=curcell->w;
      Wold.p=curcell->p;
      Wold.a=sqrt(GAMMA*Wold.p/Wold.d);
#ifdef WRADHYD
      Wold.X=curcell->X;
#endif
      dW2U(&Wold,&Uold); // primitive -> conservative

      REAL eold=Uold.eint;

      /* // MUSCL STATE RECONSTRUCTION */
      memset(ffact,0,sizeof(int)*2);

      dMUSCL_BOUND2(stencil+i, 13, icell, WC,dt,dx);// central

      for(iface=0;iface<2;iface++){
	inei=iface+2;
	memcpy(WC+iface,WC+inei,sizeof(struct Wtype)); // moving the data towards idx=0,1
	//memcpy(WC+iface,&Wold,sizeof(struct Wtype)); // moving the data towards idx=0,1

	dW2U(WC+iface,UC+iface);
      }

      // Neighbor MUSCL reconstruction
      for(iface=0;iface<2;iface++){
	inei=iface+2;
	dMUSCL_BOUND2(stencil+i, ioct[vnei[inei]], vcell[inei], WT,dt,dx);// 
	memcpy(WN+iface,WT+idxnei[inei],sizeof(struct Wtype)); 
	//memcpy(WN+iface,&(stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].field),sizeof(struct Wtype)); 

       	dW2U(WN+iface,UN+iface);
	
	if(!stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].split){
	  ffact[iface]=1; // we cancel the contriubtion of split neighbors
	}

      }




      // Y DIRECTION =========================================================================
      
      // --------- solving the Riemann Problems FRONT

      // Switching to Split description

/* 	// =========================================== */

#ifdef RIEMANN_HLLC
      dspeedestimateY_HLLC(&WN[0],&WC[0],&SL,&SR,&pstar,&ustar);

	if(SL>=0.){
	  dgetflux_Y(&UN[0],FL);
#ifdef DUAL_E
	  memcpy(&Us,&UN[0],sizeof(struct Utype));
#endif

	}
	else if(SR<=0.){
	  dgetflux_Y(&UC[0],FL);
#ifdef DUAL_E
	  memcpy(&Us,&UC[0],sizeof(struct Utype));
#endif
	}
	else if((SL<0.)&&(ustar>=0.)){
	  dgetflux_Y(&UN[0],FL);
	  fact=WN[0].d*(SL-WN[0].v)/(SL-ustar);
	  FL[0]+=(fact*1.                                                                      -UN[0].d )*SL;
	  FL[1]+=(fact*WN[0].u                                                                 -UN[0].du)*SL;
	  FL[2]+=(fact*ustar                                                                   -UN[0].dv)*SL;
	  FL[3]+=(fact*WN[0].w                                                                 -UN[0].dw)*SL;
	  FL[4]+=(fact*(UN[0].E/UN[0].d+(ustar-WN[0].v)*(ustar+WN[0].p/(WN[0].d*(SL-WN[0].v))))-UN[0].E )*SL;

#ifdef DUAL_E
	  Us.d =(fact*1.);
	  Us.du=(fact*WN[0].u);
	  Us.dv=(fact*ustar);
	  Us.dw=(fact*WN[0].w);
	  Us.E =(fact*(UN[0].E/UN[0].d+(ustar-WN[0].v)*(ustar+WN[0].p/(WN[0].d*(SL-WN[0].v)))));
#endif
#ifdef WRADHYD
	  FL[6]+=(fact*WN[0].X                                                                 -UN[0].dX)*SL;
#endif
	}
	else if((ustar<=0.)&&(SR>0.)){
	  dgetflux_Y(&UC[0],FL);
	  fact=WC[0].d*(SR-WC[0].v)/(SR-ustar);
	  FL[0]+=(fact*1.                                                                      -UC[0].d )*SR;
	  FL[1]+=(fact*WC[0].u                                                                 -UC[0].du)*SR;
	  FL[2]+=(fact*ustar                                                                   -UC[0].dv)*SR;
	  FL[3]+=(fact*WC[0].w                                                                 -UC[0].dw)*SR;
	  FL[4]+=(fact*(UC[0].E/UC[0].d+(ustar-WC[0].v)*(ustar+WC[0].p/(WC[0].d*(SR-WC[0].v))))-UC[0].E )*SR;

#ifdef DUAL_E
	  Us.d =(fact*1.);
	  Us.du=(fact*WC[0].u);
	  Us.dv=(fact*ustar);
	  Us.dw=(fact*WC[0].w);
	  Us.E =(fact*(UC[0].E/UC[0].d+(ustar-WC[0].v)*(ustar+WC[0].p/(WC[0].d*(SR-WC[0].v)))));
#endif
#ifdef WRADHYD
	  FL[6]+=(fact*WC[0].X                                                                 -UC[0].dX)*SR;
#endif
	}


#ifdef DUAL_E
	ebar=(Us.E-0.5*(Us.du*Us.du+Us.dv*Us.dv+Us.dw*Us.dw)/Us.d); 
	FL[5]=(Us.dv/Us.d*ebar);
	divuloc=(GAMMA-1.)*(Us.dv/Us.d)*eold;
	divu+=-divuloc;
#endif


#endif
	// ===========================================




      // --------- solving the Riemann Problems BACK


      // Switching to Split description

	//=====================================================

#ifdef RIEMANN_HLLC
	dspeedestimateY_HLLC(&WC[1],&WN[1],&SL,&SR,&pstar,&ustar);

	if(SL>=0.){
	  dgetflux_Y(&UC[1],FR);
#ifdef DUAL_E
	  memcpy(&Us,&UC[1],sizeof(struct Utype));
#endif

	}
	else if(SR<=0.){
	  dgetflux_Y(&UN[1],FR);
#ifdef DUAL_E
	  memcpy(&Us,&UN[1],sizeof(struct Utype));
#endif

	}
	else if((SL<0.)&&(ustar>=0.)){
	  dgetflux_Y(&UC[1],FR);
	  fact=WC[1].d*(SL-WC[1].v)/(SL-ustar);
	  FR[0]+=(fact*1.                                                                      -UC[1].d )*SL;
	  FR[1]+=(fact*WC[1].u                                                                 -UC[1].du)*SL;
	  FR[2]+=(fact*ustar                                                                   -UC[1].dv)*SL;
	  FR[3]+=(fact*WC[1].w                                                                 -UC[1].dw)*SL;
	  FR[4]+=(fact*(UC[1].E/UC[1].d+(ustar-WC[1].v)*(ustar+WC[1].p/(WC[1].d*(SL-WC[1].v))))-UC[1].E )*SL;

#ifdef DUAL_E
	  Us.d =(fact*1.);
	  Us.du=(fact*WC[1].u);
	  Us.dv=(fact*ustar);
	  Us.dw=(fact*WC[1].w);
	  Us.E =(fact*(UC[1].E/UC[1].d+(ustar-WC[1].v)*(ustar+WC[1].p/(WC[1].d*(SL-WC[1].v)))));
#endif
#ifdef WRADHYD
	  FR[6]+=(fact*WC[1].X                                                                 -UC[1].dX)*SL;
#endif
	}
	else if((ustar<=0.)&&(SR>0.)){
	  dgetflux_Y(&UN[1],FR);
	  fact=WN[1].d*(SR-WN[1].v)/(SR-ustar);
	  FR[0]+=(fact*1.                                                                      -UN[1].d )*SR;
	  FR[1]+=(fact*WN[1].u                                                                 -UN[1].du)*SR;
	  FR[2]+=(fact*ustar                                                                   -UN[1].dv)*SR;
	  FR[3]+=(fact*WN[1].w                                                                 -UN[1].dw)*SR;
	  FR[4]+=(fact*(UN[1].E/UN[1].d+(ustar-WN[1].v)*(ustar+WN[1].p/(WN[1].d*(SR-WN[1].v))))-UN[1].E )*SR;

#ifdef DUAL_E
	  Us.d =(fact*1.);
	  Us.du=(fact*WN[1].u);
	  Us.dv=(fact*ustar);
	  Us.dw=(fact*WN[1].w);
	  Us.E =(fact*(UN[1].E/UN[1].d+(ustar-WN[1].v)*(ustar+WN[1].p/(WN[1].d*(SR-WN[1].v)))));
#endif
#ifdef WRADHYD
	  FR[6]+=(fact*WN[1].X                                                                 -UN[1].dX)*SR;
#endif
	}


#ifdef DUAL_E
	ebar=(Us.E-0.5*(Us.du*Us.du+Us.dv*Us.dv+Us.dw*Us.dw)/Us.d); 
	divuloc=(GAMMA-1.)*(Us.dv/Us.d)*eold;
	FR[5]=(Us.dv/Us.d*ebar);
	divu+= divuloc;
#endif

#endif

      
      //========================= copy the fluxes
      // Cancelling the fluxes from splitted neighbours
	
	for(iface=0;iface<NVAR;iface++) FL[iface]*=ffact[0]; 
	for(iface=0;iface<NVAR;iface++) FR[iface]*=ffact[1]; 
	
	memcpy(stencil[i].New.cell[icell].flux+2*NVAR,FL,sizeof(REAL)*NVAR);
	memcpy(stencil[i].New.cell[icell].flux+3*NVAR,FR,sizeof(REAL)*NVAR);
	
	stencil[i].New.cell[icell].divu=divu;
	
    //ready for the next oct
  }
  }
}

//===================================================================================================
//===================================================================================================

__global__ void dhydroM_sweepX(struct HGRID *stencil, int nread,REAL dx, REAL dt){

  int inei,icell,iface;
  int i;
  int vnei[6],vcell[6];

  REAL FL[NVAR],FR[NVAR];
  struct Utype Uold;
  struct Wtype Wold;
  REAL pstar,ustar;

  struct Wtype WT[6]; // FOR MUSCL RECONSTRUCTION
  struct Wtype WC[6]; // FOR MUSCL RECONSTRUCTION

  struct Utype UC[2];
  struct Utype UN[2];
  struct Wtype WN[2];

  int ioct[7]={12,14,10,16,4,22,13};
  int idxnei[6]={1,0,3,2,5,4};

  struct Wtype *curcell;

  REAL SL,SR;
  
  int ffact[2]={0,0};
  REAL fact;

#ifdef DUAL_E
  struct Utype Us;
  REAL ebar;
  REAL divu,divuloc;
#endif
  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;
  
  i=bx*blockDim.x+tx;
  if(i<nread){
  for(icell=0;icell<8;icell++){ // we scan the cells
    getcellnei_gpu_hydro(icell, vnei, vcell); // we get the neighbors
      
  
      
      memset(FL,0,sizeof(REAL)*NVAR);
      memset(FR,0,sizeof(REAL)*NVAR);

      // Getting the original state ===========================
      
      curcell=&(stencil[i].oct[ioct[6]].cell[icell].field);
      
#ifdef DUAL_E
      divu=stencil[i].New.cell[icell].divu;
#endif      

      Wold.d=curcell->d;
      Wold.u=curcell->u;;
      Wold.v=curcell->v;;
      Wold.w=curcell->w;;
      Wold.p=curcell->p;;
      Wold.a=sqrt(GAMMA*Wold.p/Wold.d);
#ifdef WRADHYD
      Wold.X=curcell->X;
#endif
      dW2U(&Wold,&Uold); // primitive -> conservative
      REAL eold=Uold.eint;

      /* // MUSCL STATE RECONSTRUCTION */
      memset(ffact,0,sizeof(int)*2);

      dMUSCL_BOUND2(stencil+i, 13, icell, WC,dt,dx);// central
      for(iface=0;iface<2;iface++){
	dW2U(WC+iface,UC+iface);
      }

      // Neighbor MUSCL reconstruction
      for(iface=0;iface<2;iface++){
	inei=iface;
	dMUSCL_BOUND2(stencil+i, ioct[vnei[inei]], vcell[inei], WT,dt,dx);// 
	memcpy(WN+iface,WT+idxnei[inei],sizeof(struct Wtype)); 
       	dW2U(WN+iface,UN+iface);
	
	if(!stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].split){
	  ffact[iface]=1; // we cancel the contriubtion of split neighbors
	}

      }




      // X DIRECTION =========================================================================
      
      // --------- solving the Riemann Problems LEFT

      // Switching to Split description

/* 	// =========================================== */

#ifdef RIEMANN_HLLC
      dspeedestimateX_HLLC(&WN[0],&WC[0],&SL,&SR,&pstar,&ustar);

      if(SL>=0.){
	dgetflux_X(&UN[0],FL);
#ifdef DUAL_E
	memcpy(&Us,&UN[0],sizeof(struct Utype));
#endif

	}
      else if(SR<=0.){
	dgetflux_X(&UC[0],FL);
#ifdef DUAL_E
	memcpy(&Us,&UC[0],sizeof(struct Utype));
#endif
	}
      else if((SL<0.)&&(ustar>=0.)){
	dgetflux_X(&UN[0],FL);
	fact=WN[0].d*(SL-WN[0].u)/(SL-ustar);
	FL[0]+=(fact*1.                                                                      -UN[0].d )*SL;
	FL[1]+=(fact*ustar                                                                   -UN[0].du)*SL;
	FL[2]+=(fact*WN[0].v                                                                 -UN[0].dv)*SL;
	FL[3]+=(fact*WN[0].w                                                                 -UN[0].dw)*SL;
	FL[4]+=(fact*(UN[0].E/UN[0].d+(ustar-WN[0].u)*(ustar+WN[0].p/(WN[0].d*(SL-WN[0].u))))-UN[0].E )*SL;

#ifdef DUAL_E
	  Us.d =(fact*1.);
	  Us.du=(fact*ustar);
	  Us.dv=(fact*WN[0].v);
	  Us.dw=(fact*WN[0].w);
	  Us.E =(fact*(UN[0].E/UN[0].d+(ustar-WN[0].u)*(ustar+WN[0].p/(WN[0].d*(SL-WN[0].u)))));
#endif
#ifdef WRADHYD
	 FL[6]+=(fact*WN[0].X                                                                 -UN[0].dX)*SL;
#endif
	}
      else if((ustar<=0.)&&(SR>0.)){
	dgetflux_X(&UC[0],FL);
	  fact=WC[0].d*(SR-WC[0].u)/(SR-ustar);
	  FL[0]+=(fact*1.                                                                      -UC[0].d )*SR;
	  FL[1]+=(fact*ustar                                                                   -UC[0].du)*SR;
	  FL[2]+=(fact*WC[0].v                                                                 -UC[0].dv)*SR;
	  FL[3]+=(fact*WC[0].w                                                                 -UC[0].dw)*SR;
	  FL[4]+=(fact*(UC[0].E/UC[0].d+(ustar-WC[0].u)*(ustar+WC[0].p/(WC[0].d*(SR-WC[0].u))))-UC[0].E )*SR;

#ifdef DUAL_E
	  Us.d =(fact*1.);
	  Us.du=(fact*ustar);
	  Us.dv=(fact*WC[0].v);
	  Us.dw=(fact*WC[0].w);
	  Us.E =(fact*(UC[0].E/UC[0].d+(ustar-WC[0].u)*(ustar+WC[0].p/(WC[0].d*(SR-WC[0].u)))));
#endif
#ifdef WRADHYD
	  FL[6]+=(fact*WC[0].X                                                                 -UC[0].dX)*SR;
#endif
	}


#ifdef DUAL_E
      ebar=(Us.E-0.5*(Us.du*Us.du+Us.dv*Us.dv+Us.dw*Us.dw)/Us.d); 
      divuloc=(GAMMA-1.)*(Us.du/Us.d)*eold;
      FL[5]=(Us.du/Us.d*ebar);
      divu+=-divuloc;
#endif


#endif
	

	// ===========================================


      

      // --------- solving the Riemann Problems RIGHT


      // Switching to Split description

	//=====================================================

#ifdef RIEMANN_HLLC
      dspeedestimateX_HLLC(&WC[1],&WN[1],&SL,&SR,&pstar,&ustar);

	if(SL>=0.){
	  dgetflux_X(&UC[1],FR);
#ifdef DUAL_E
	  memcpy(&Us,&UC[1],sizeof(struct Utype));
#endif

	}
	else if(SR<=0.){
	  dgetflux_X(&UN[1],FR);
#ifdef DUAL_E
	  memcpy(&Us,&UN[1],sizeof(struct Utype));
#endif

	}
	else if((SL<0.)&&(ustar>=0.)){
	  dgetflux_X(&UC[1],FR);
	  fact=WC[1].d*(SL-WC[1].u)/(SL-ustar);
	  FR[0]+=(fact*1.                                                                      -UC[1].d )*SL;
	  FR[1]+=(fact*ustar                                                                   -UC[1].du)*SL;
	  FR[2]+=(fact*WC[1].v                                                                 -UC[1].dv)*SL;
	  FR[3]+=(fact*WC[1].w                                                                 -UC[1].dw)*SL;
	  FR[4]+=(fact*(UC[1].E/UC[1].d+(ustar-WC[1].u)*(ustar+WC[1].p/(WC[1].d*(SL-WC[1].u))))-UC[1].E )*SL;

#ifdef DUAL_E
	  Us.d =(fact*1.);
	  Us.du=(fact*ustar);
	  Us.dv=(fact*WC[1].v);
	  Us.dw=(fact*WC[1].w);
	  Us.E =(fact*(UC[1].E/UC[1].d+(ustar-WC[1].u)*(ustar+WC[1].p/(WC[1].d*(SL-WC[1].u)))));
#endif
#ifdef WRADHYD
	  FR[6]+=(fact*WC[1].X                                                                 -UC[1].dX)*SL;
#endif
	}
	else if((ustar<=0.)&&(SR>0.)){
	  dgetflux_X(&UN[1],FR);
	  fact=WN[1].d*(SR-WN[1].u)/(SR-ustar);
	  FR[0]+=(fact*1.                                                                      -UN[1].d )*SR;
	  FR[1]+=(fact*ustar                                                                   -UN[1].du)*SR;
	  FR[2]+=(fact*WN[1].v                                                                 -UN[1].dv)*SR;
	  FR[3]+=(fact*WN[1].w                                                                 -UN[1].dw)*SR;
	  FR[4]+=(fact*(UN[1].E/UN[1].d+(ustar-WN[1].u)*(ustar+WN[1].p/(WN[1].d*(SR-WN[1].u))))-UN[1].E )*SR;

#ifdef DUAL_E
	  Us.d =(fact*1.);
	  Us.du=(fact*ustar);
	  Us.dv=(fact*WN[1].v);
	  Us.dw=(fact*WN[1].w);
	  Us.E =(fact*(UN[1].E/UN[1].d+(ustar-WN[1].u)*(ustar+WN[1].p/(WN[1].d*(SR-WN[1].u)))));
#endif
#ifdef WRADHYD
	  FR[6]+=(fact*WN[1].X                                                                 -UN[1].dX)*SR;
#endif
	}

#ifdef DUAL_E
      ebar=(Us.E-0.5*(Us.du*Us.du+Us.dv*Us.dv+Us.dw*Us.dw)/Us.d); 
      divuloc=(GAMMA-1.)*(Us.du/Us.d)*eold;
      FR[5]=(Us.du/Us.d*ebar);
      divu+= divuloc;
#endif

#endif





      
      //========================= copy the fluxes
      // Cancelling the fluxes from splitted neighbours

      for(iface=0;iface<NVAR;iface++) FL[iface]*=ffact[0]; 
      for(iface=0;iface<NVAR;iface++) FR[iface]*=ffact[1]; 

      memcpy(stencil[i].New.cell[icell].flux+0*NVAR,FL,sizeof(REAL)*NVAR);
      memcpy(stencil[i].New.cell[icell].flux+1*NVAR,FR,sizeof(REAL)*NVAR);

      stencil[i].New.cell[icell].divu=divu;

    //ready for the next oct
  }
  }
}


// ==============================================================================================================
// ==============================================================================================================

__global__ void dupdatefield(struct HGRID *stencil, int nread, int stride, struct CPUINFO *cpu, REAL dxcur, REAL dtnew)
{
  int i,icell;
  struct Utype U;
  REAL one;
  int flx;
  REAL dtsurdx=dtnew/dxcur;
  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;
  i=bx*blockDim.x+tx;
  if(i<nread){
  for(icell=0;icell<8;icell++){ // we scan the cells
      
    if(stencil[i].oct[13].cell[icell].split) continue;
    
    // ==== updating
    // actually we compute and store the delta U only
    one=1.;
    memset(&U,0,sizeof(struct Utype)); // setting delta U
    for(flx=0;flx<6;flx++){
      U.d +=stencil[i].New.cell[icell].flux[0+flx*NVAR]*dtsurdx*one;
      U.du+=stencil[i].New.cell[icell].flux[1+flx*NVAR]*dtsurdx*one;
      U.dv+=stencil[i].New.cell[icell].flux[2+flx*NVAR]*dtsurdx*one;
      U.dw+=stencil[i].New.cell[icell].flux[3+flx*NVAR]*dtsurdx*one;
      U.E +=stencil[i].New.cell[icell].flux[4+flx*NVAR]*dtsurdx*one;
#ifdef DUAL_E
      U.eint+=stencil[i].New.cell[icell].flux[5+flx*NVAR]*dtsurdx*one;
#endif
#ifdef WRADHYD
	U.dX+=stencil[i].New.cell[icell].flux[6+flx*NVAR]*dtsurdx*one;
#endif
      one*=-1.;
    }
    // scatter back the delta Uwithin the stencil
    
    memcpy(&(stencil[i].New.cell[icell].deltaU),&U,sizeof(struct Utype));

  }
  }
}

// =======================================================

//=======================================================================
//=======================================================================

int advancehydroGPU(struct OCT **firstoct, int level, struct CPUINFO *cpu, struct HGRID *stencil, int stride, REAL dxcur, REAL dtnew){

  struct OCT *nextoct;
  struct OCT *curoct;
  int nreadtot,nread;
  double t[10];
  double tg=0.,th=0.,tu=0.,ts=0.;
  int is;
  int offset;
  
  // --------------- setting the first oct of the level
  nextoct=firstoct[level-1];
  nreadtot=0;
  int ng;
  int nt;

  cudaStream_t stream[cpu->nstream];

  // creating the streams
  for(is=0;is<cpu->nstream;is++){
    cudaStreamCreate(&stream[is]);
  }
  
  // Calculations

  if((nextoct!=NULL)&&(cpu->noct[level-1]!=0)){
    do {
      curoct=nextoct;
      nextoct=curoct->next; 

      t[0]=MPI_Wtime();
  
      // ------------ gathering the stencil value values
      nextoct= gatherstencil(curoct,stencil,stride,cpu, &nread);
      ng=((nread-1)/cpu->nthread/cpu->nstream)+1; // +1 is for leftovers

      if(ng==1){
	nt=nread;
      }
      else{
	nt=cpu->nthread;
      }

      /* dim3 gridoct((nread/cpu->nthread/cpu->nstream)>1?(nread/cpu->nthread/cpu->nstream):1); */
      /* dim3 blockoct(cpu->nthread); */
      dim3 gridoct(ng);
      dim3 blockoct(nt);

      // streaming ====================
      for(is=0;is<cpu->nstream;is++){
	/* cudaDeviceSynchronize(); */
	/* t[2]=MPI_Wtime(); */
	
	offset=is*nread/cpu->nstream;
	//printf("Start Error Hyd =%s\n",cudaGetErrorString(cudaGetLastError()));
	cudaMemcpyAsync(cpu->hyd_stencil+offset,stencil+offset,nread*sizeof(struct HGRID)/cpu->nstream,cudaMemcpyHostToDevice,stream[is]);  
	//printf("Start Error Hyd =%s\n",cudaGetErrorString(cudaGetLastError()));
      
	// ------------ solving the hydro
	dhydroM_sweepX<<<gridoct,blockoct,0,stream[is]>>>(cpu->hyd_stencil+offset,nread,dxcur,dtnew);
	//printf("Start Error =%s\n",cudaGetErrorString(cudaGetLastError()));
	dhydroM_sweepY<<<gridoct,blockoct,0,stream[is]>>>(cpu->hyd_stencil+offset,nread,dxcur,dtnew); 
	//printf("Start Error =%s\n",cudaGetErrorString(cudaGetLastError()));
	dhydroM_sweepZ<<<gridoct,blockoct,0,stream[is]>>>(cpu->hyd_stencil+offset,nread,dxcur,dtnew); 
	//printf("Start Error =%s\n",cudaGetErrorString(cudaGetLastError()));
	
	// ------------ updating values within the stencil
	/* cudaDeviceSynchronize(); */
	/* t[4]=MPI_Wtime(); */

	dupdatefield<<<gridoct,blockoct,0,stream[is]>>>(cpu->hyd_stencil+offset,nread,stride,cpu,dxcur,dtnew);
	//printf("Start Error =%s\n",cudaGetErrorString(cudaGetLastError()));

	/* cudaDeviceSynchronize(); */
	/* t[6]=MPI_Wtime(); */
	
	cudaMemcpyAsync(stencil+offset,cpu->hyd_stencil+offset,nread/cpu->nstream*sizeof(struct HGRID),cudaMemcpyDeviceToHost,stream[is]);  
	//printf("Start Error =%s\n",cudaGetErrorString(cudaGetLastError()));
      }
      /* dim3 gridoct2((nread/cpu->nthread)>1?(nread/cpu->nthread):1); */
      /* dim3 blockoct2(cpu->nthread); */

      
      /* dev_updatefield<<<gridoct2,blockoct2>>>(cpu->hyd_stencil,nread,stride,cpu,dxcur,dtnew); */
      
      cudaDeviceSynchronize();
      // ------------ scatter back the FLUXES
      //cudaMemcpy(stencil,cpu->hyd_stencil,nread*sizeof(struct HGRID),cudaMemcpyDeviceToHost);  
      nextoct=scatterstencil(curoct,stencil, nread, cpu,dxcur,dtnew);
      
      t[8]=MPI_Wtime();
      
      nreadtot+=nread;
      
      
      ts+=(t[8]-t[6]);
      tu+=(t[6]-t[4]);
      th+=(t[4]-t[2]);
      tg+=(t[2]-t[0]);
    }while(nextoct!=NULL);
  }
  //printf("GPU | tgat=%e tcal=%e tup=%e tscat=%e\n",tg,th,tu,ts);

  // Destroying the streams
  for(is=0;is<cpu->nstream;is++){
    cudaStreamDestroy(stream[is]);
  }


  return nreadtot;
}

#endif